#include "hip/hip_runtime.h"
#include "layer_norm.h"
#include "src/lyradiff/utils/cuda_type_utils.cuh"
#include <stdio.h>
namespace lyradiff {

constexpr int kWarpSize = 32;

template<typename T>
inline __device__ void WelfordCombine(T val, T* mean, T* m2, T* count)
{
    // Use Welford Online algorithem to compute mean and variance
    // For more details you can refer to:
    // https://en.wikipedia.org/wiki/Algorithms_for_calculating_variance#Welford's_online_algorithm
    *count += 1;
    T delta1 = val - *mean;
    *mean += delta1 / *count;
    T delta2 = val - *mean;
    *m2 += delta1 * delta2;
}

template<typename T>
inline __device__ void WelfordCombine(T b_mean, T b_m2, T b_count, T* mean, T* m2, T* count)
{
    if (b_count == cuda_cast<T>(0.0)) {
        return;
    }
    T new_count = *count + b_count;
    T nb_over_n = b_count / new_count;
    T delta     = b_mean - *mean;
    *mean += delta * nb_over_n;
    *m2 += b_m2 + delta * delta * (*count) * nb_over_n;
    *count = new_count;
}

template<typename T, int thread_group_width = kWarpSize>
__inline__ __device__ void WelfordWarpReduce(T thread_mean, T thread_m2, T thread_count, T* mean, T* m2, T* count)
{
    *mean  = thread_mean;
    *m2    = thread_m2;
    *count = thread_count;
    for (int mask = thread_group_width / 2; mask > 0; mask /= 2) {
        T b_mean  = __shfl_down_sync(0xffffffff, *mean, mask, thread_group_width);
        T b_m2    = __shfl_down_sync(0xffffffff, *m2, mask, thread_group_width);
        T b_count = __shfl_down_sync(0xffffffff, *count, mask, thread_group_width);
        WelfordCombine(b_mean, b_m2, b_count, mean, m2, count);
    }
}

template<typename T>
__inline__ __device__ void
WelfordBlockAllReduce(T thread_mean, T thread_m2, T thread_count, T* result_mean, T* result_m2, T* result_count)
{
    __shared__ T mean_shared[kWarpSize];
    __shared__ T m2_shared[kWarpSize];
    __shared__ T count_shared[kWarpSize];
    __shared__ T mean_result_broadcast;
    __shared__ T m2_result_broadcast;
    __shared__ T count_result_broadcast;
    const int    lid        = threadIdx.x % kWarpSize;
    const int    wid        = threadIdx.x / kWarpSize;
    T            warp_mean  = 0;
    T            warp_m2    = 0;
    T            warp_count = 0;
    WelfordWarpReduce(thread_mean, thread_m2, thread_count, &warp_mean, &warp_m2, &warp_count);
    __syncthreads();
    if (lid == 0) {
        mean_shared[wid]  = warp_mean;
        m2_shared[wid]    = warp_m2;
        count_shared[wid] = warp_count;
    }
    __syncthreads();
    if (wid == 0) {
        if (threadIdx.x < blockDim.x / kWarpSize) {
            warp_mean  = mean_shared[lid];
            warp_m2    = m2_shared[lid];
            warp_count = count_shared[lid];
        }
        else {
            warp_mean  = static_cast<T>(0);
            warp_m2    = static_cast<T>(0);
            warp_count = static_cast<T>(0);
        }
        __syncwarp();
        T block_mean  = 0;
        T block_m2    = 0;
        T block_count = 0;
        WelfordWarpReduce(warp_mean, warp_m2, warp_count, &block_mean, &block_m2, &block_count);
        if (lid == 0) {
            mean_result_broadcast  = block_mean;
            m2_result_broadcast    = block_m2;
            count_result_broadcast = block_count;
        }
    }
    __syncthreads();
    *result_mean  = mean_result_broadcast;
    *result_m2    = m2_result_broadcast;
    *result_count = count_result_broadcast;
}

template<typename T, int32_t cols_per_thread>
__global__ void LayerNormBlockSMemImpl(
    T* dst, const T* src, const T* gamma, const T* beta, const double epsilon, const int32_t rows, const int32_t cols)
{
    // extern __shared__ __align__(sizeof(double)) unsigned char shared_buf[];
    // auto*                                                     buf = reinterpret_cast<T*>(shared_buf);

    int       row_id = blockIdx.x;
    const int tid    = threadIdx.x;

    constexpr int32_t num_iters_per_thread = cols_per_thread / 2;

    using T2 = typename TypeConverter<T>::Type;  // float to float2, half to half2
    __align__(2 * sizeof(T)) T2 buf[num_iters_per_thread];
    __align__(2 * sizeof(T)) T2 gamma_buf[num_iters_per_thread];
    __align__(2 * sizeof(T)) T2 beta_buf[num_iters_per_thread];
    T                           thread_mean  = 0.0;
    T                           thread_m2    = 0.0;
    T                           delta        = 0.0;
    T                           delta2       = 0.0;
    T                           thread_count = 0.0;

    // 加载每个线程的数据到寄存器
#pragma unroll
    for (int32_t i = 0; i < num_iters_per_thread; ++i) {
        int32_t offset = row_id * cols + tid * cols_per_thread + i * 2;
        T2      ele2   = *reinterpret_cast<const T2*>(src + offset);
        // 第一个元素 welford
        thread_count += 1;
        delta = ele2.x - thread_mean;
        thread_mean += delta / thread_count;
        delta2 = ele2.x - thread_mean;
        thread_m2 += delta * delta2;

        // 第二个元素 welford
        thread_count += 1;
        delta = ele2.y - thread_mean;
        thread_mean += delta / thread_count;
        delta2 = ele2.y - thread_mean;
        thread_m2 += delta * delta2;

        buf[i] = ele2;

        int32_t gb_offset = tid * cols_per_thread + i * 2;
        gamma_buf[i]      = *reinterpret_cast<const T2*>(gamma + gb_offset);
        beta_buf[i]       = *reinterpret_cast<const T2*>(beta + gb_offset);
    }

    // 线程束之间每个线程使用 welford 算法聚合 mean, m2, 以及元素数量 count

    T row_mean  = 0;
    T row_m2    = 0;
    T row_count = 0;

    WelfordBlockAllReduce<T>(thread_mean, thread_m2, thread_count, &row_mean, &row_m2, &row_count);

    // T row_mean     = warp_mean;
    T row_variance = cuda_max<T>(row_m2 / static_cast<T>(row_count), static_cast<T>(0.0));
    T row_inv_var  = rsqrt(row_variance + static_cast<T>(epsilon));

#pragma unroll
    for (int32_t i = 0; i < num_iters_per_thread; ++i) {
        // 寄存器上数据算数操作
        T2 ele2 = buf[i];
        T2 g2   = gamma_buf[i];
        T2 b2   = beta_buf[i];

        ele2.x = (ele2.x - row_mean) * row_inv_var;
        ele2.y = (ele2.y - row_mean) * row_inv_var;

        ele2.x = g2.x * ele2.x + b2.x;
        ele2.y = g2.y * ele2.y + b2.y;

        int32_t offset = row_id * cols + tid * cols_per_thread + i * 2;
        // 会写全局显存变量
        *reinterpret_cast<T2*>(&dst[offset]) = ele2;
    }
}


template<typename T, int32_t cols_per_thread>
__global__ void warpLayerNorm(
    T* dst, const T* src, const T* gamma, const T* beta, const double epsilon, const int32_t rows, const int32_t cols)
{
    // 一个线程束，最大分配 32 个线程
    // 一个线程在 A100 上可最大获得 255 bytes 的寄存器:
    //  分配 32 个线程， 每个线程最大可获得 255bytes 寄存器，half2 类型，每个线程一次处理 2 个元素，4个字节
    // 当 cols = 320 时：
    //      需迭代 320/32/2=5 次，寄存器 5*4 = 20 个bytes
    // 当 cols = 640 时：
    //      需迭代 640/32/2=10 次，寄存器 10*4 = 40 个bytes
    // 当 cols = 1024 时：
    //      需迭代 1024/32/2=16 次，寄存器 16*4 = 64 个bytes
    // 当 cols = 1280 时：
    //      需迭代 1280/32/2=20 次，寄存器 20*4 = 80 个bytes  （但我不确定目前这个尺寸是warp上做更快还是 block 上更快）

    // 数据布局：[B, C, NHiddens] ---> [B*C, NHiddens] ---> [Rows, Cols] ---> Flatten
    // gamma, beta: [NHiddens]
    // Grid: [B, C]
    // Block: [32,]
    constexpr int32_t num_iters_per_thread = cols_per_thread / 2;

    using T2 = typename TypeConverter<T>::Type;  // float to float2, half to half2
    __align__(2 * sizeof(T)) T2 buf[num_iters_per_thread];
    __align__(2 * sizeof(T)) T2 gamma_buf[num_iters_per_thread];
    __align__(2 * sizeof(T)) T2 beta_buf[num_iters_per_thread];
    T                           thread_mean  = 0.0;
    T                           thread_m2    = 0.0;
    T                           delta        = 0.0;
    T                           delta2       = 0.0;
    T                           thread_count = 0.0;

    // int row_id = blockIdx.x * gridDim.y + blockIdx.y;
    int row_id = blockIdx.x;

    int tid = threadIdx.x;

// 加载每个线程的数据到寄存器
#pragma unroll
    for (int32_t i = 0; i < num_iters_per_thread; ++i) {
        int32_t offset = row_id * cols + tid * cols_per_thread + i * 2;
        T2      ele2   = *reinterpret_cast<const T2*>(src + offset);
        // 第一个元素 welford
        thread_count += 1;
        delta = ele2.x - thread_mean;
        thread_mean += delta / thread_count;
        delta2 = ele2.x - thread_mean;
        thread_m2 += delta * delta2;

        // 第二个元素 welford
        thread_count += 1;
        delta = ele2.y - thread_mean;
        thread_mean += delta / thread_count;
        delta2 = ele2.y - thread_mean;
        thread_m2 += delta * delta2;

        buf[i] = ele2;

        int32_t gb_offset = tid * cols_per_thread + i * 2;
        gamma_buf[i]      = *reinterpret_cast<const T2*>(gamma + gb_offset);
        beta_buf[i]       = *reinterpret_cast<const T2*>(beta + gb_offset);
    }

    // 线程束之间每个线程使用 welford 算法聚合 mean, m2, 以及元素数量 count
    T warp_mean, warp_m2;
    T warp_count;
    warp_mean  = thread_mean;
    warp_m2    = thread_m2;
    warp_count = thread_count;

#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        T b_mean  = __shfl_down_sync(0xffffffff, warp_mean, mask, 32);
        T b_m2    = __shfl_down_sync(0xffffffff, warp_m2, mask, 32);
        T b_count = __shfl_down_sync(0xffffffff, warp_count, mask, 32);

        T new_count = warp_count + b_count;
        T nb_over_n = b_count / new_count;
        T delta     = b_mean - warp_mean;
        warp_mean += delta * nb_over_n;
        warp_m2 += b_m2 + delta * delta * warp_count * nb_over_n;
        warp_count = new_count;
    }

    // welford 聚合求得的值广播到每个线程
    warp_mean  = __shfl_sync(0xffffffff, warp_mean, 0, 32);
    warp_m2    = __shfl_sync(0xffffffff, warp_m2, 0, 32);
    warp_count = __shfl_sync(0xffffffff, warp_count, 0, 32);

    T row_mean     = warp_mean;
    T row_variance = cuda_max<T>(warp_m2 / static_cast<T>(warp_count), static_cast<T>(0.0));
    T row_inv_var  = rsqrt(row_variance + static_cast<T>(epsilon));

#pragma unroll
    for (int32_t i = 0; i < num_iters_per_thread; ++i) {
        // 寄存器上数据算数操作
        T2 ele2 = buf[i];
        T2 g2   = gamma_buf[i];
        T2 b2   = beta_buf[i];

        ele2.x = (ele2.x - row_mean) * row_inv_var;
        ele2.y = (ele2.y - row_mean) * row_inv_var;

        ele2.x = g2.x * ele2.x + b2.x;
        ele2.y = g2.y * ele2.y + b2.y;

        int32_t offset = row_id * cols + tid * cols_per_thread + i * 2;
        // 会写全局显存变量
        *reinterpret_cast<T2*>(&dst[offset]) = ele2;
    }
}

template<typename T, int32_t cols_per_thread>
__global__ void warpLayerNormWithShiftAndScale(
    T* dst, const T* src, const T* scale, const T* shift, const double epsilon, const int32_t rows, const int32_t cols)
{
    // 一个线程束，最大分配 32 个线程
    // 一个线程在 A100 上可最大获得 255 bytes 的寄存器:
    //  分配 32 个线程， 每个线程最大可获得 255bytes 寄存器，half2 类型，每个线程一次处理 2 个元素，4个字节
    // 当 cols = 320 时：
    //      需迭代 320/32/2=5 次，寄存器 5*4 = 20 个bytes
    // 当 cols = 640 时：
    //      需迭代 640/32/2=10 次，寄存器 10*4 = 40 个bytes
    // 当 cols = 1024 时：
    //      需迭代 1024/32/2=16 次，寄存器 16*4 = 64 个bytes
    // 当 cols = 1280 时：
    //      需迭代 1280/32/2=20 次，寄存器 20*4 = 80 个bytes  （但我不确定目前这个尺寸是warp上做更快还是 block 上更快）

    // 数据布局：[B, C, NHiddens] ---> [B*C, NHiddens] ---> [Rows, Cols] ---> Flatten
    // scale, shift: [B, NHiddens]
    // Grid: [B, C]
    // Block: [32,]
    constexpr int32_t num_iters_per_thread = cols_per_thread / 2;

    using T2 = typename TypeConverter<T>::Type;  // float to float2, half to half2
    __align__(2 * sizeof(T)) T2 buf[num_iters_per_thread];
    __align__(2 * sizeof(T)) T2 shift_buf[num_iters_per_thread];
    __align__(2 * sizeof(T)) T2 scale_buf[num_iters_per_thread];
    T                           thread_mean  = 0.0;
    T                           thread_m2    = 0.0;
    T                           delta        = 0.0;
    T                           delta2       = 0.0;
    T                           thread_count = 0.0;
    T                           one          = 1.0;

    // int row_id = blockIdx.x * gridDim.y + blockIdx.y;
    int row_id = blockIdx.x + blockIdx.y * rows;

    int tid = threadIdx.x;

// 加载每个线程的数据到寄存器
#pragma unroll
    for (int32_t i = 0; i < num_iters_per_thread; ++i) {
        int32_t offset = row_id * cols + tid * cols_per_thread + i * 2;
        T2      ele2   = *reinterpret_cast<const T2*>(src + offset);
        // 第一个元素 welford
        thread_count += 1;
        delta = ele2.x - thread_mean;
        thread_mean += delta / thread_count;
        delta2 = ele2.x - thread_mean;
        thread_m2 += delta * delta2;

        // 第二个元素 welford
        thread_count += 1;
        delta = ele2.y - thread_mean;
        thread_mean += delta / thread_count;
        delta2 = ele2.y - thread_mean;
        thread_m2 += delta * delta2;

        buf[i] = ele2;

        int32_t gb_offset = blockIdx.y * cols + tid * cols_per_thread + i * 2;
        scale_buf[i]      = *reinterpret_cast<const T2*>(scale + gb_offset);
        shift_buf[i]      = *reinterpret_cast<const T2*>(shift + gb_offset);
    }

    // 线程束之间每个线程使用 welford 算法聚合 mean, m2, 以及元素数量 count
    T warp_mean, warp_m2;
    T warp_count;
    warp_mean  = thread_mean;
    warp_m2    = thread_m2;
    warp_count = thread_count;

#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        T b_mean  = __shfl_down_sync(0xffffffff, warp_mean, mask, 32);
        T b_m2    = __shfl_down_sync(0xffffffff, warp_m2, mask, 32);
        T b_count = __shfl_down_sync(0xffffffff, warp_count, mask, 32);

        T new_count = warp_count + b_count;
        T nb_over_n = b_count / new_count;
        T delta     = b_mean - warp_mean;
        warp_mean += delta * nb_over_n;
        warp_m2 += b_m2 + delta * delta * warp_count * nb_over_n;
        warp_count = new_count;
    }

    // welford 聚合求得的值广播到每个线程
    warp_mean  = __shfl_sync(0xffffffff, warp_mean, 0, 32);
    warp_m2    = __shfl_sync(0xffffffff, warp_m2, 0, 32);
    warp_count = __shfl_sync(0xffffffff, warp_count, 0, 32);

    T row_mean     = warp_mean;
    T row_variance = cuda_max<T>(warp_m2 / static_cast<T>(warp_count), static_cast<T>(0.0));
    T row_inv_var  = rsqrt(row_variance + static_cast<T>(epsilon));

#pragma unroll
    for (int32_t i = 0; i < num_iters_per_thread; ++i) {
        // 寄存器上数据算数操作
        T2 ele2   = buf[i];
        T2 scale2 = scale_buf[i];
        T2 shift2 = shift_buf[i];

        ele2.x = (ele2.x - row_mean) * row_inv_var;
        ele2.y = (ele2.y - row_mean) * row_inv_var;

        ele2.x = (scale2.x + one) * ele2.x + shift2.x;
        ele2.y = (scale2.y + one) * ele2.y + shift2.y;

        int32_t offset = row_id * cols + tid * cols_per_thread + i * 2;
        // 会写全局显存变量
        *reinterpret_cast<T2*>(&dst[offset]) = ele2;
    }
}

template<typename T, int32_t cols_per_thread>
__global__ void LayerNormBlockWithShiftAndScaleSMemImpl(
    T* dst, const T* src, const T* scale, const T* shift, const double epsilon, const int32_t rows, const int32_t cols)
{
    // extern __shared__ __align__(sizeof(double)) unsigned char shared_buf[];
    // auto*                                                     buf = reinterpret_cast<T*>(shared_buf);
    // scale, shift: [B, NHiddens]
    int row_id = blockIdx.x + blockIdx.y * rows;
    int tid    = threadIdx.x;

    constexpr int32_t num_iters_per_thread = cols_per_thread / 2;

    using T2 = typename TypeConverter<T>::Type;  // float to float2, half to half2
    __align__(2 * sizeof(float)) float2 buf[num_iters_per_thread];
    __align__(2 * sizeof(float)) T2     shift_buf[num_iters_per_thread];
    __align__(2 * sizeof(float)) T2     scale_buf[num_iters_per_thread];
    float                               thread_mean  = 0.0;
    float                               thread_m2    = 0.0;
    float                               delta        = 0.0;
    float                               delta2       = 0.0;
    float                               thread_count = 0.0;
    T                                   one          = 1.0;
    // 加载每个线程的数据到寄存器
#pragma unroll
    for (int32_t i = 0; i < num_iters_per_thread; ++i) {
        int32_t offset = row_id * cols + tid * cols_per_thread + i * 2;
        T2      tmp2   = *reinterpret_cast<const T2*>(src + offset);
        float2  ele2   = cuda_cast<float2>(tmp2);
        // 第一个元素 welford
        thread_count += 1;
        delta = ele2.x - thread_mean;
        thread_mean += delta / thread_count;
        delta2 = ele2.x - thread_mean;
        thread_m2 += delta * delta2;

        // 第二个元素 welford
        thread_count += 1;
        delta = ele2.y - thread_mean;
        thread_mean += delta / thread_count;
        delta2 = ele2.y - thread_mean;
        thread_m2 += delta * delta2;

        buf[i] = ele2;

        int32_t gb_offset = blockIdx.y * cols + tid * cols_per_thread + i * 2;
        scale_buf[i]      = *reinterpret_cast<const T2*>(scale + gb_offset);
        shift_buf[i]      = *reinterpret_cast<const T2*>(shift + gb_offset);
    }

    // 线程束之间每个线程使用 welford 算法聚合 mean, m2, 以及元素数量 count

    float row_mean  = 0;
    float row_m2    = 0;
    float row_count = 0;

    WelfordBlockAllReduce<float>(thread_mean, thread_m2, thread_count, &row_mean, &row_m2, &row_count);

    // T row_mean     = warp_mean;
    float row_variance = cuda_max<float>(row_m2 / static_cast<float>(row_count), static_cast<float>(0.0));
    float row_inv_var  = rsqrt(row_variance + static_cast<float>(epsilon));

#pragma unroll
    for (int32_t i = 0; i < num_iters_per_thread; ++i) {
        // 寄存器上数据算数操作
        float2 ele2   = buf[i];
        T2     scale2 = scale_buf[i];
        T2     shift2 = shift_buf[i];

        ele2.x  = (ele2.x - row_mean) * row_inv_var;
        ele2.y  = (ele2.y - row_mean) * row_inv_var;
        T2 tmp2 = cuda_cast<T2>(ele2);

        tmp2.x = (scale2.x + one) * tmp2.x + shift2.x;
        tmp2.y = (scale2.y + one) * tmp2.y + shift2.y;

        int32_t offset = row_id * cols + tid * cols_per_thread + i * 2;
        // 会写全局显存变量
        *reinterpret_cast<T2*>(&dst[offset]) = tmp2;
    }
}

template<typename T>
void invokeLayerNorm(T*           dst,
                     const T*     src,
                     const T*     gamma,
                     const T*     beta,
                     size_t       batch_size,
                     size_t       channels,
                     size_t       nhiddens,
                     hipStream_t stream,
                     const double eps)
{
    const int32_t rows = batch_size * channels;
    const int32_t cols = nhiddens;

    // dim3 grid(batch_size, channels);
    // printf("layernorm nhiddens %d\n", nhiddens);

    dim3 grid(rows);

    dim3 block(32);
    switch (cols) {
        case 64 /* constant-expression */:
            /* code */
            warpLayerNorm<T, 2><<<grid, block, 0, stream>>>(dst, src, gamma, beta, eps, rows, cols);
            break;

        case 320 /* constant-expression */:
            /* code */
            warpLayerNorm<T, 10><<<grid, block, 0, stream>>>(dst, src, gamma, beta, eps, rows, cols);
            break;

        case 640:
            warpLayerNorm<T, 20><<<grid, block, 0, stream>>>(dst, src, gamma, beta, eps, rows, cols);
            break;

        case 1024:
            warpLayerNorm<T, 32><<<grid, block, 0, stream>>>(dst, src, gamma, beta, eps, rows, cols);
            break;

        case 1280:  // 1280 目前使用 LayerNormBlockSMemImpl
            // warpLayerNorm<T, 40><<<grid, block, 0, stream>>>(dst, src, gamma, beta, eps, rows, cols);
            LayerNormBlockSMemImpl<T, 10><<<grid, 128, 0, stream>>>(dst, src, gamma, beta, eps, rows, cols);
            break;

        case 768:
            warpLayerNorm<T, 24><<<grid, block, 0, stream>>>(dst, src, gamma, beta, eps, rows, cols);
            break;

        default:
            break;
    }
}

template<typename T>
void invokeLayerNormWithShiftAndScale(T*           dst,
                                      const T*     src,
                                      const T*     scale,
                                      const T*     shift,
                                      size_t       batch_size,
                                      size_t       channels,
                                      size_t       nhiddens,
                                      hipStream_t stream,
                                      const double eps)
{
    const int32_t rows = channels;
    const int32_t cols = nhiddens;

    // dim3 grid(batch_size, channels);

    dim3 grid(rows, batch_size);

    dim3 block(32);
    switch (cols) {
        case 64 /* constant-expression */:
            /* code */
            warpLayerNormWithShiftAndScale<T, 2><<<grid, block, 0, stream>>>(dst, src, scale, shift, eps, rows, cols);
            break;

        case 320 /* constant-expression */:
            /* code */
            warpLayerNormWithShiftAndScale<T, 10><<<grid, block, 0, stream>>>(dst, src, scale, shift, eps, rows, cols);
            break;

        case 640:
            warpLayerNormWithShiftAndScale<T, 20><<<grid, block, 0, stream>>>(dst, src, scale, shift, eps, rows, cols);
            break;

        case 1024:
            warpLayerNormWithShiftAndScale<T, 32><<<grid, block, 0, stream>>>(dst, src, scale, shift, eps, rows, cols);
            break;

        case 1280:  // 1280 目前使用 LayerNormBlockSMemImpl
            // warpLayerNorm<T, 40><<<grid, block, 0, stream>>>(dst, src, gamma, beta, eps, rows, cols);
            LayerNormBlockWithShiftAndScaleSMemImpl<T, 10>
                <<<grid, 128, 0, stream>>>(dst, src, scale, shift, eps, rows, cols);
            break;
        case 3072:  // 3072 目前使用 LayerNormBlockSMemImpl
            // warpLayerNorm<T, 40><<<grid, block, 0, stream>>>(dst, src, gamma, beta, eps, rows, cols);
            LayerNormBlockWithShiftAndScaleSMemImpl<T, 24>
                <<<grid, 128, 0, stream>>>(dst, src, scale, shift, eps, rows, cols);
            break;
        case 768:
            warpLayerNormWithShiftAndScale<T, 24><<<grid, block, 0, stream>>>(dst, src, scale, shift, eps, rows, cols);
            break;

        default:
            break;
    }
}

// 为 float 和 half 做模板特化 (BCN)
#define INSTANTIATE_INVOKE_LAYER_NORM(T)                                                                               \
    template void invokeLayerNorm(T*           dst,                                                                    \
                                  const T*     src,                                                                    \
                                  const T*     gamma,                                                                  \
                                  const T*     beta,                                                                   \
                                  size_t       batch_size,                                                             \
                                  size_t       channels,                                                               \
                                  size_t       nhiddens,                                                               \
                                  hipStream_t stream,                                                                 \
                                  const double eps)

INSTANTIATE_INVOKE_LAYER_NORM(float);
INSTANTIATE_INVOKE_LAYER_NORM(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_LAYER_NORM(__hip_bfloat16);
#endif
#undef INSTANTIATE_INVOKE_LAYER_NORM

// 为 float 和 half 做模板特化 (BCN)
#define INSTANTIATE_INVOKE_LAYER_NORM_WITH_SHIFT_AND_SCALE(T)                                                          \
    template void invokeLayerNormWithShiftAndScale(T*           dst,                                                   \
                                                   const T*     src,                                                   \
                                                   const T*     scale,                                                 \
                                                   const T*     shift,                                                 \
                                                   size_t       batch_size,                                            \
                                                   size_t       channels,                                              \
                                                   size_t       nhiddens,                                              \
                                                   hipStream_t stream,                                                \
                                                   const double eps)

INSTANTIATE_INVOKE_LAYER_NORM_WITH_SHIFT_AND_SCALE(float);
INSTANTIATE_INVOKE_LAYER_NORM_WITH_SHIFT_AND_SCALE(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_LAYER_NORM_WITH_SHIFT_AND_SCALE(__hip_bfloat16);
#endif
#undef INSTANTIATE_INVOKE_LAYER_NORM_WITH_SHIFT_AND_SCALE

}  // namespace lyradiff