#include "hip/hip_runtime.h"


#include "src/lyradiff/kernels/cutlass_kernels/fpA_intB_gemm/fpA_intB_gemm_template.h"

namespace lyradiff
{
namespace kernels
{
namespace cutlass_kernels
{
#ifdef ENABLE_FP8
template class CutlassFpAIntBGemmRunner<__hip_fp8_e4m3_fnuz,       
    cutlass::int4b_t,                                        
    cutlass::WeightOnlyQuantOp::PER_COLUMN_SCALE_ONLY, half, 
    half,                                                    
    half                                                     
    >;
#endif
} // namespace cutlass_kernels
} // namespace kernels
} // namespace lyradiff
